#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

// If want to print the number array and get the max of sequential, then let the ISSQUENTIAL be 'true'
#define ISSQUENTIAL false   
// If want to check the answer of GPU method is correct or not, let the CHECKANS be 'true'
#define CHECKANS true
#define THREAD 1024

// Global functions
unsigned int getmax(unsigned int *, unsigned int);
unsigned int getmaxcu(unsigned int *, unsigned int);
__global__ void getMaximum(unsigned int num[], unsigned int size);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array

    if(argc !=2)
    {
        printf("usage: maxseq num\n");
        printf("num = size of the array\n");
        exit(1);
    }

    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
        printf("Unable to allocate mem for an array of size %u\n", size);
        exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
        numbers[i] = rand()  % size;    

    if( ISSQUENTIAL ){
        for(int i = 0; i < size; i++)   printf("%d ", numbers[i]);  // This is the array print out
        printf("\n (Sequential) The maximum number in the array is: %u\n", 
                getmax(numbers, size));
    }

    // GPU method
    printf(" The maximum number in the array is: %u\n", 
            getmaxcu(numbers, size));

    free(numbers);
    exit(0);
}


/*
input: pointer to an array of long int
number of elements in the array
output: the maximum number of the array
 */
unsigned int getmax(unsigned int num[], unsigned int size)
{
    unsigned int i;
    unsigned int max = num[0];

    for(i = 1; i < size; i++)
        if(num[i] > max)
            max = num[i];

    return( max );
}

/*
    input: pointer to an array of unsigned int
        number of elements in the array
    output: the maxumum number of the array
 */
unsigned int getmaxcu(unsigned int number[], unsigned int n)
{
    // setup GPU
    int deviceId = 0;
    hipError_t error = hipGetDevice(&deviceId);
    hipSetDevice(1);

    // error handling
    if( error != hipSuccess )
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    unsigned int size = ceil(n / THREAD);
    unsigned int maxval = 0;
    unsigned int threads = n;
    if(n > THREAD)  threads = THREAD;

    unsigned int blocks = (n + threads - 1) / threads;
    unsigned int *num, *answer;
    unsigned int lenNum = n * sizeof(unsigned int);
    unsigned int lenBlock = blocks * sizeof(unsigned int);

    // Transfer to device
    hipMalloc((void **)&num, lenNum);
    hipMemcpy(num, number, lenNum, hipMemcpyHostToDevice);

    // Kernal invocation for 
    hipDeviceSynchronize();
    getMaximum<<<1, 1>>>(num, n);

    // Transfer to host
    answer = (unsigned int *)malloc(lenBlock);
    hipMemcpy(answer, num, lenBlock, hipMemcpyDeviceToHost);

    maxval = answer[0];

    // Free
    hipFree(num);
    free(answer);
    // All data to be flushed
    hipDeviceReset();

    if( CHECKANS )
    {
        unsigned int seq = getmax(number, n);
        if(seq != maxval)
            printf("The answer is incorrect: (sequential)%d (GPU)%d\n", seq, maxval);
    }
    return maxval;
}

__global__ void getMaximum(unsigned int num[], unsigned int size)
{
    for(int i=0; i<size; i++)
    {
        if(num[i] > num[0])
            num[0] = num[i];
    }
}
