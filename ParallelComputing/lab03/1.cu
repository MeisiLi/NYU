#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

#define ISSQUENTIAL false
#define THREAD 1024

unsigned int getmax(unsigned int *, unsigned int);
unsigned int getmaxcu(unsigned int *, unsigned int);
__global__ void getMaximum(unsigned int num[], unsigned int size);
__global__ void getMax(unsigned int num[], unsigned int size, unsigned int ans[]);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array

    if(argc !=2)
    {
        printf("usage: maxseq num\n");
        printf("num = size of the array\n");
        exit(1);
    }

    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
        printf("Unable to allocate mem for an array of size %u\n", size);
        exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
        numbers[i] = rand()  % size;    

    if( ISSQUENTIAL ){
        for(int i = 0; i < size; i++)   printf("%d ", numbers[i]);
        printf("\n (Sequential) The maximum number in the array is: %u\n", 
                getmax(numbers, size));
    }

    
    printf(" (GPU) The maximum number in the array is: %u\n", 
            getmaxcu(numbers, size));

    free(numbers);
    exit(0);
}


/*
input: pointer to an array of long int
number of elements in the array
output: the maximum number of the array
 */
unsigned int getmax(unsigned int num[], unsigned int size)
{
    unsigned int i;
    unsigned int max = num[0];

    for(i = 1; i < size; i++)
        if(num[i] > max)
            max = num[i];

    return( max );

}

unsigned int getmaxcu(unsigned int number[], unsigned int n)
{
    // setup GPU
    int deviceId = 0;
    hipError_t error = hipGetDevice(&deviceId);;

    hipSetDevice(1);

    // error handling
    if( error != hipSuccess )
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    unsigned int size = ceil(n / THREAD);
    unsigned int maxval, threads;
    if(n > THREAD)  threads = THREAD;
    else    threads = n;

    unsigned int blocks = (n + threads - 1) / threads;
    unsigned int *result, *num, *answer;
    unsigned int len = n * sizeof(unsigned int);
    unsigned int lenBlock = blocks * sizeof(unsigned int);

    // Transfer to device
    hipMalloc((void **)&num, len);
    hipMemcpy(num, number, len, 
        hipMemcpyHostToDevice);

    hipMalloc((void **)&result, len);
    //printf("len: %d lenBlock: %d\n", len, lenBlock);
    hipMemcpy(result, number, len, hipMemcpyHostToDevice);

    // Kernal invocation for 
    hipDeviceSynchronize();
    getMaximum<<<1, 1>>>(num, n);

    // Transfer to host
    answer = (unsigned int *)malloc(lenBlock);
    hipMemcpy(answer, num, lenBlock, 
        hipMemcpyDeviceToHost);

    maxval = answer[0];

    hipFree(num);
    hipFree(result);
    free(answer);

    hipDeviceReset();

    return maxval;
}

__global__ void getMaximum(unsigned int num[], unsigned int size)
{
    for(int i=0; i<size; i++)
    {
        if(num[i] > num[0])
            num[0] = num[i];
    }
}

__global__ void getMax(unsigned int num[], unsigned int size, unsigned int ans[])
{
    __shared__ unsigned int shared[THREAD];

    // global thread id 
    int grid = blockIdx.x * blockDim.x + threadIdx.x;
    
    shared[threadIdx.x] = num[grid];
    if( threadIdx.x >= size )   shared[threadIdx.x] = 0;

    __syncthreads();

    ans[threadIdx.x] = shared[grid];

    printf("grid: %d, threadIdx.x: %d blockDim.x: %d\n", 
        grid, threadIdx.x, blockDim.x);

    for(unsigned int s = blockDim.x; s > 0; s = s / 2)
    {
        int mid = s / 2;
        if(threadIdx.x < mid)
            if(ans[threadIdx.x] < ans[threadIdx.x + mid])
                ans[threadIdx.x] = ans[threadIdx.x + mid];
        __syncthreads();
    }

    /*
    if(threadIdx.x == 0)
        printf("block: %d\n", ans[0]);
    */

    ans[blockIdx.x] = shared[0];
}
